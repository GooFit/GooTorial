#include "hip/hip_runtime.h"
#include "Variable.hh" 
#include "GaussianThrustFunctor.hh" 
#include "AddThrustFunctor.hh" 
#include "PolynomialThrustFunctor.hh" 
#include "PdfBuilder.hh" 
#include "UnbinnedDataSet.hh" 

#include "TRandom.hh" 
#include "TH1F.h"
#include "TH2F.h"
#include "TStyle.h" 
#include "TCanvas.h" 

#include <sys/time.h>
#include <sys/times.h>
#include <iostream>

using namespace std; 

int main (int argc, char** argv) {
  gStyle->SetCanvasBorderMode(0);
  gStyle->SetCanvasColor(10);
  gStyle->SetFrameFillColor(10);
  gStyle->SetFrameBorderMode(0);
  gStyle->SetPadColor(0);
  gStyle->SetTitleColor(1);
  gStyle->SetStatColor(0);
  gStyle->SetFillColor(0);
  gStyle->SetFuncWidth(1);
  gStyle->SetLineWidth(1);
  gStyle->SetLineColor(1);
  gStyle->SetPalette(1, 0);

  vector<Variable*> vars; 
  Variable* xvar = new Variable("xvar", -5, 5); vars.push_back(xvar);
  UnbinnedDataSet data(vars);

  TH1F xvarHist("xvarHist", "", 
		xvar->numbins, xvar->lowerlimit, xvar->upperlimit);

  xvarHist.SetStats(false); 

  TRandom donram(42); 
  double totalData = 0; 
  for (int i = 0; i < 100000; ++i) {
    xvar->value = donram.Gaus(0.2, 1.1);
    if (donram.Uniform() < 0.1) xvar->value = donram.Uniform(xvar->lowerlimit, xvar->upperlimit); 
    if (fabs(xvar->value) > 5) {--i; continue;} 
    data.addEvent(); 
    xvarHist.Fill(xvar->value);
    totalData++; 
  }

  Variable* xmean = new Variable("xmean", 0, 1, -10, 10);
  Variable* xsigm = new Variable("xsigm", 1, 0.5, 1.5); 
  GaussianThrustFunctor signal("signal", xvar, xmean, xsigm); 

  vars.clear();
  Variable* constant = new Variable("constant", 1.0); 
  vars.push_back(constant); 
  PolynomialThrustFunctor backgr("backgr", xvar, vars); 

  vector<FunctorBase*> comps;
  comps.push_back(&signal);
  comps.push_back(&backgr);

  vars.clear(); 
  Variable* sigFrac = new Variable("sigFrac", 0.9, 0.75, 1.00); 
  vars.push_back(sigFrac); 

  AddThrustFunctor total("total", vars, comps);
  total.setData(&data);
  PdfFunctor fitter(&total);
  fitter.fit(); 
  fitter.getMinuitValues(); 
 
  TH1F pdfHist("pdfHist", "", 
		xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  TH1F sigHist("sigHist", "", 
		xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  TH1F bkgHist("bkgHist", "", 
		xvar->numbins, xvar->lowerlimit, xvar->upperlimit);

  pdfHist.SetStats(false);
  sigHist.SetStats(false);
  bkgHist.SetStats(false);

  UnbinnedDataSet grid(xvar);
  for (int i = 0; i < xvar->numbins; ++i) {
    double step = (xvar->upperlimit - xvar->lowerlimit)/xvar->numbins;
    xvar->value = xvar->lowerlimit + (i + 0.5) * step;
    grid.addEvent(); 
  }

  total.setData(&grid);
  vector<vector<double> > pdfVals;
  total.getCompProbsAtDataPoints(pdfVals); 

  TCanvas foo;

  double totalPdf = 0; 
  for (int i = 0; i < grid.getNumEvents(); ++i) {
    grid.loadEvent(i); 
    pdfHist.Fill(xvar->value, pdfVals[0][i]);
    sigHist.Fill(xvar->value, pdfVals[1][i]);
    bkgHist.Fill(xvar->value, pdfVals[2][i]);
    totalPdf += pdfVals[0][i]; 
  }

  for (int i = 0; i < xvar->numbins; ++i) {
    double val = pdfHist.GetBinContent(i+1); 
    val /= totalPdf; 
    val *= totalData;
    pdfHist.SetBinContent(i+1, val); 
    val = sigHist.GetBinContent(i+1); 
    val /= totalPdf; 
    val *= sigFrac->value; 
    val *= totalData;
    sigHist.SetBinContent(i+1, val); 
    val = bkgHist.GetBinContent(i+1); 
    val /= totalPdf; 
    val *= (1.0 - sigFrac->value);
    val *= totalData;
    bkgHist.SetBinContent(i+1, val); 
  }

  xvarHist.SetMarkerStyle(8);
  xvarHist.SetMarkerSize(0.5);
  xvarHist.Draw("p"); 
  pdfHist.SetLineColor(kBlue);
  pdfHist.SetLineWidth(3); 
  pdfHist.Draw("lsame"); 
  sigHist.SetLineColor(kBlue);
  sigHist.SetLineStyle(kDashed); 
  sigHist.SetLineWidth(3); 
  sigHist.Draw("lsame"); 
  bkgHist.SetLineColor(kRed);
  bkgHist.SetLineWidth(3); 
  bkgHist.Draw("lsame"); 
  foo.SaveAs("xhist.png"); 


  
  return 0;
}
