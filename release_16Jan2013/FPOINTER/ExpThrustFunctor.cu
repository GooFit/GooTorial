#include "ExpThrustFunctor.hh"

__device__ fptype device_Exp (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x = evt[indices[2 + indices[0]]]; 
  fptype alpha = p[indices[1]];

  fptype ret = EXP(alpha*x); 
  return ret; 
}

__device__ fptype device_ExpOffset (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x = evt[indices[2 + indices[0]]]; 
  x -= indices[1]; 
  fptype alpha = p[indices[2]];

  fptype ret = EXP(alpha*x); 
  return ret; 
}

__device__ fptype device_ExpPoly (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x = evt[indices[2 + indices[0]]]; 
  
  fptype exparg = 0; 
  for (int i = 0; i <= indices[0]; ++i) {
    exparg += POW(x, i) * p[indices[i+1]]; 
  }
  
  fptype ret = EXP(exparg); 
  return ret; 
}

__device__ fptype device_ExpPolyOffset (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x = evt[indices[2 + indices[0]]]; 
  x -= indices[1]; 
  
  fptype exparg = 0; 
  for (int i = 0; i <= indices[0]; ++i) {
    exparg += POW(x, i) * p[indices[i+2]]; 
  }
  
  fptype ret = EXP(exparg); 
  return ret; 
}

__device__ device_function_ptr ptr_to_Exp = device_Exp; 
__device__ device_function_ptr ptr_to_ExpPoly = device_ExpPoly; 
__device__ device_function_ptr ptr_to_ExpOffset = device_ExpOffset; 
__device__ device_function_ptr ptr_to_ExpPolyOffset = device_ExpPolyOffset; 

__host__ ExpThrustFunctor::ExpThrustFunctor (std::string n, Variable* _x, Variable* alpha, Variable* offset) 
  : ThrustPdfFunctor(_x, n) 
{
  std::vector<unsigned int> pindices;
  if (offset) {
    pindices.push_back(registerParameter(offset));
    pindices.push_back(registerParameter(alpha));
    hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_ExpOffset, sizeof(void*));
    initialise(pindices); 
  }
  else {
    pindices.push_back(registerParameter(alpha));
    hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_Exp, sizeof(void*));
    initialise(pindices); 
  }
}

__host__ ExpThrustFunctor::ExpThrustFunctor (std::string n, Variable* _x, std::vector<Variable*>& weights, Variable* offset) 
  : ThrustPdfFunctor(_x, n) 
{
  std::vector<unsigned int> pindices;
  if (offset) pindices.push_back(registerParameter(offset)); 
  assert(0 < weights.size()); 
  for (std::vector<Variable*>::iterator w = weights.begin(); w != weights.end(); ++w) {
    pindices.push_back(registerParameter(*w)); 
  }
  if (offset) hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_ExpPolyOffset, sizeof(void*));
  else hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_ExpPoly, sizeof(void*));
  initialise(pindices); 
}

__host__ fptype ExpThrustFunctor::integrate (fptype lo, fptype hi) const {
  fptype alpha = host_params[host_indices[parameters + 1]]; 

  if (0 == alpha) {
    // This gives a constant 1 all across the range
    return (hi - lo); 
  }

  fptype ret = EXP(alpha*hi) - EXP(alpha*lo);
  ret /= alpha; 
  return ret; 
}

