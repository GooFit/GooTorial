#include "hip/hip_runtime.h"
#include "../GlobalCudaDefines.hh"
#include "ThrustPdfFunctor.hh" 
#include "thrust/sequence.h" 
#include "thrust/iterator/constant_iterator.h" 

//#ifdef CUDAPRINT
#include "cuPrintf.cu" 
#include <fstream> 
//#endif 

// These variables are either function-pointer related (thus specific to this implementation)
// or constrained to be in the CUDAglob translation unit by nvcc limitations; otherwise they 
// would be in FunctorBase. 

// Device-side, translation-unit constrained. 
__constant__ fptype hipArray[maxParams];           // Holds device-side fit parameters. 
__constant__ unsigned int paramIndices[maxParams];  // Holds functor-specific indices into hipArray. Also overloaded to hold integer constants (ie parameters that cannot vary.) 
__constant__ fptype functorConstants[maxParams];    // Holds non-integer constants. Notice that first entry is number of events. 
__constant__ fptype normalisationFactors[maxParams]; 

// For debugging 
__constant__ int callnumber; 
__constant__ int gpuDebug; 
__constant__ unsigned int debugParamIndex;
__device__ int internalDebug1 = -1; 
__device__ int internalDebug2 = -1; 
__device__ int internalDebug3 = -1; 
int cpuDebug = 0; 

// Function-pointer related. 
__device__ void* device_function_table[200]; // Not clear why this cannot be __constant__, but it causes crashes to declare it so. 
void* host_function_table[200];
unsigned int num_device_functions = 0; 
#ifdef OMP_ON
// Make functionAddressToDevideIndexMap and array of maps indexed by thread id since 
// I get the following compiler error if I try to make it threadprivate.
// "functionAddressToDeviceIndexMap’ declared ‘threadprivate’ after first use"
typedef std::map<void*, int> tMapType;
tMapType functionAddressToDeviceIndexMap[MAX_THREADS]; 
#pragma omp threadprivate(host_function_table, num_device_functions)
fptype gSum;
fptype sums[MAX_THREADS];
double gLognorm;
double lognorms[MAX_THREADS];
#else
std::map<void*, int> functionAddressToDeviceIndexMap; 
#endif


#define cutilSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)

// For use in debugging memory issues
void printMemoryStatus (std::string file, int line) {
  size_t memfree = 0;
  size_t memtotal = 0; 
  hipDeviceSynchronize(); 
  hipMemGetInfo(&memfree, &memtotal); 
  hipDeviceSynchronize(); 
  std::cout << "Memory status " << file << " " << line << " Free " << memfree << " Total " << memtotal << " Used " << (memtotal - memfree) << std::endl;
}


#include <execinfo.h>
void* stackarray[10];
void abortWithCudaPrintFlush (std::string file, int line, std::string reason, const FunctorBase* pdf = 0) {
#ifdef CUDAPRINT
  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();
#endif
  std::cout << "Abort called from " << file << " line " << line << " due to " << reason << std::endl; 
  if (pdf) {
    std::set<Variable*> pars;
    pdf->getParameters(pars);
    std::cout << "Parameters of " << pdf->getName() << " : \n";
    for (std::set<Variable*>::iterator v = pars.begin(); v != pars.end(); ++v) {
      if (0 > (*v)->index) continue; 
      std::cout << "  " << (*v)->name << " (" << (*v)->index << ") :\t" << host_params[(*v)->index] << std::endl;
    }
  }

  std::cout << "Parameters (" << totalParams << ") :\n"; 
  for (int i = 0; i < totalParams; ++i) {
    std::cout << host_params[i] << " ";
  }
  std::cout << std::endl; 


  // get void* pointers for all entries on the stack
  size_t size = backtrace(stackarray, 10);
  // print out all the frames to stderr
  backtrace_symbols_fd(stackarray, size, 2);

  exit(1); 
}

void __cudaSafeCall (hipError_t err, const char* file, int line) {
  if (hipSuccess != err) {
    std::cout << "Error code " << err << " (" << hipGetErrorString(err) << ") at " << file << ", " << line << std::endl;
    exit(1); 
  }
}

__device__ fptype calculateEval (fptype rawPdf, fptype* evtVal, unsigned int par) {
  // Just return the raw PDF value, for use in (eg) normalisation. 
  return rawPdf; 
}

__device__ fptype calculateNLL (fptype rawPdf, fptype* evtVal, unsigned int par) {
  //if ((10 > callnumber) && (threadIdx.x < 10) && (blockIdx.x == 0)) cuPrintf("calculateNll %i %f %f %f\n", callnumber, rawPdf, normalisationFactors[par], rawPdf*normalisationFactors[par]);
  rawPdf *= normalisationFactors[par];
  return rawPdf > 0 ? -LOG(rawPdf) : 0; 
}

__device__ fptype calculateProb (fptype rawPdf, fptype* evtVal, unsigned int par) {
  // Return probability, ie normalised PDF value.
  return rawPdf * normalisationFactors[par];
}

__device__ fptype calculateBinAvg (fptype rawPdf, fptype* evtVal, unsigned int par) {
  rawPdf *= normalisationFactors[par];
  rawPdf *= evtVal[1]; // Bin volume 
  // Log-likelihood of numEvents with expectation of exp is (-exp + numEvents*ln(exp) - ln(numEvents!)). 
  // The last is constant, so we drop it; and then multiply by minus one to get the negative log-likelihood. 
  if (rawPdf > 0) {
    fptype expEvents = functorConstants[0]*rawPdf;
    return (expEvents - evtVal[0]*log(expEvents)); 
  }
  return 0; 
}

__device__ fptype calculateBinWithError (fptype rawPdf, fptype* evtVal, unsigned int par) {
  // In this case interpret the rawPdf as just a number, not a number of events. 
  // Do not divide by integral over phase space, do not multiply by bin volume, 
  // and do not collect 200 dollars. evtVal should have the structure (bin entry, bin error). 
  //printf("[%i, %i] ((%f - %f) / %f)^2 = %f\n", blockIdx.x, threadIdx.x, rawPdf, evtVal[0], evtVal[1], POW((rawPdf - evtVal[0]) / evtVal[1], 2)); 
  rawPdf -= evtVal[0]; // Subtract observed value.
  rawPdf /= evtVal[1]; // Divide by error.
  rawPdf *= rawPdf; 
  return rawPdf; 
}

__device__ fptype calculateChisq (fptype rawPdf, fptype* evtVal, unsigned int par) {
  rawPdf *= normalisationFactors[par];
  rawPdf *= evtVal[1]; // Bin volume 

  return pow(rawPdf * functorConstants[0] - evtVal[0], 2) / (evtVal[0] > 1 ? evtVal[0] : 1); 
}

__device__ device_metric_ptr ptr_to_Eval         = calculateEval; 
__device__ device_metric_ptr ptr_to_NLL          = calculateNLL;  
__device__ device_metric_ptr ptr_to_Prob         = calculateProb; 
__device__ device_metric_ptr ptr_to_BinAvg       = calculateBinAvg;  
__device__ device_metric_ptr ptr_to_BinWithError = calculateBinWithError;
__device__ device_metric_ptr ptr_to_Chisq        = calculateChisq; 

void* host_fcn_ptr = 0;

void* getMetricPointer (std::string name) {
#define CHOOSE_PTR(ptrname) if (name == #ptrname) hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptrname, sizeof(void*))
  host_fcn_ptr = 0; 
  CHOOSE_PTR(ptr_to_Eval); 
  CHOOSE_PTR(ptr_to_NLL); 
  CHOOSE_PTR(ptr_to_Prob); 
  CHOOSE_PTR(ptr_to_BinAvg); 
  CHOOSE_PTR(ptr_to_BinWithError); 
  CHOOSE_PTR(ptr_to_Chisq); 

  assert(host_fcn_ptr); 

  return host_fcn_ptr;
#undef CHOOSE_PTR
}


ThrustPdfFunctor::ThrustPdfFunctor (Variable* x, std::string n) 
  : FunctorBase(x, n)
  , logger(0)
{}

__host__ int ThrustPdfFunctor::findFunctionIdx (void* dev_functionPtr) {
  // Code specific to function-pointer implementation 
#ifdef OMP_ON
  int tid = omp_get_thread_num();
  std::map<void*, int>::iterator localPos = functionAddressToDeviceIndexMap[tid].find(dev_functionPtr); // Use find instead of [] to avoid returning 0 if the index doesn't exist.
  if (localPos != functionAddressToDeviceIndexMap[tid].end()) {
    return (*localPos).second; 
  }
#else
  std::map<void*, int>::iterator localPos = functionAddressToDeviceIndexMap.find(dev_functionPtr); 
  if (localPos != functionAddressToDeviceIndexMap.end()) {
    return (*localPos).second; 
  }
#endif

  int fIdx = num_device_functions;   
  host_function_table[num_device_functions] = dev_functionPtr;
#ifdef OMP_ON 
  functionAddressToDeviceIndexMap[tid][dev_functionPtr] = num_device_functions; 
#else
  functionAddressToDeviceIndexMap[dev_functionPtr] = num_device_functions; 
#endif
  num_device_functions++; 
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(device_function_table), host_function_table, num_device_functions*sizeof(void*))); 
  return fIdx; 
}

__host__ void ThrustPdfFunctor::initialise (std::vector<unsigned int> pindices, void* dev_functionPtr) {
  if (!fitControl) setFitControl(new UnbinnedNllFit()); 

  // MetricTaker must be created after FunctorBase initialisation is done.
  FunctorBase::initialiseIndices(pindices); 

  functionIdx = findFunctionIdx(dev_functionPtr); 
  setMetrics(); 
}

__host__ void ThrustPdfFunctor::setDebugMask (int mask, bool setSpecific) const {
  cpuDebug = mask; 
  hipMemcpyToSymbol(HIP_SYMBOL(gpuDebug), &cpuDebug, sizeof(int), 0, hipMemcpyHostToDevice);
  if (setSpecific) hipMemcpyToSymbol(HIP_SYMBOL(debugParamIndex), &parameters, sizeof(unsigned int), 0, hipMemcpyHostToDevice);
} 

__host__ void ThrustPdfFunctor::setMetrics () {
  if (logger) delete logger;
  logger = new MetricTaker(this, getMetricPointer(fitControl->getMetric()));  
}

__host__ double ThrustPdfFunctor::sumOfNll (int numVars) const {
  static thrust::plus<double> cudaPlus;
  thrust::constant_iterator<int> eventSize(numVars); 
  thrust::constant_iterator<fptype*> arrayAddress(cudaDataArray); 
  double dummy = 0;

  //if (host_callnumber >= 2) abortWithCudaPrintFlush(__FILE__, __LINE__, getName() + " debug abort", this); 

#ifdef OMP_ON
  unsigned int thFirstEntry, thLastEntry;
  int tid, nthreads;
  int j;

  tid = omp_get_thread_num();
  nthreads = omp_get_num_threads();

  thFirstEntry = tid*(numEntries)/nthreads;
  thLastEntry = (tid+1)*(numEntries)/nthreads;

//  std::cout << tid << ": " << numEntries << " " << thFirstEntry << " " << thLastEntry << std::endl;
//  std::cout << "Extended term: " << numVars << " " << numEntries << " " << numEvents << std::endl;
    thrust::counting_iterator<int> eventIndex(0); 
    lognorms[tid] = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(eventIndex + thFirstEntry, arrayAddress, eventSize)),
					thrust::make_zip_iterator(thrust::make_tuple(eventIndex + thLastEntry, arrayAddress, eventSize)),
					*logger, dummy, cudaPlus); 
  #pragma omp barrier
  if (tid == 0) 
  {
    gLognorm = 0;
    for (j = 0; j < nthreads; j++) gLognorm += lognorms[j];
  }

  #pragma omp barrier
//  std::cout << tid << ": Full NLL: " << ret << " " << gLognorm << " " << lognorm << std::endl;
  return  gLognorm;

#else
  thrust::counting_iterator<int> eventIndex(0); 
  return thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
				  thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
				  *logger, dummy, cudaPlus);   
#endif
}

__host__ double ThrustPdfFunctor::calculateNLL () const {
  //if (cpuDebug & 1) std::cout << getName() << " entering calculateNLL" << std::endl; 

  //int oldMask = cpuDebug; 
  //if (0 == host_callnumber) setDebugMask(0, false); 
  normalise();
  //if ((0 == host_callnumber) && (1 == oldMask)) setDebugMask(1, false); 

  /*
  if (cpuDebug & 1) {
    std::cout << "Norm factors: ";
    for (int i = 0; i < totalParams; ++i) std::cout << host_normalisation[i] << " ";
    std::cout << std::endl;
  } 
  */ 
  
  hipMemcpyToSymbol(HIP_SYMBOL(normalisationFactors), host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 
  hipDeviceSynchronize(); // Ensure normalisation integrals are finished

  int numVars = observables.size(); 
  if (fitControl->binnedFit()) {
    numVars += 2;
    numVars *= -1; 
  }

  fptype ret = sumOfNll(numVars); 
  if (0 == ret) abortWithCudaPrintFlush(__FILE__, __LINE__, getName() + " zero NLL", this); 
  //if (cpuDebug & 1) std::cout << "Full NLL " << host_callnumber << " : " << 2*ret << std::endl;
  //setDebugMask(0); 

  //if ((cpuDebug & 1) && (host_callnumber >= 1)) abortWithCudaPrintFlush(__FILE__, __LINE__, getName() + " debug abort", this); 
  return 2*ret; 
}

__host__ void ThrustPdfFunctor::evaluateAtPoints (Variable* var, std::vector<fptype>& res) {
  // NB: This does not project correctly in multidimensional datasets, because all observables
  // other than 'var' will have, for every event, whatever value they happened to get set to last
  // time they were set. This is likely to be the value from the last event in whatever dataset
  // you were fitting to, but at any rate you don't get the probability-weighted integral over
  // the other observables. 

  copyParams(); 
  normalise(); 
  hipMemcpyToSymbol(HIP_SYMBOL(normalisationFactors), host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 
  UnbinnedDataSet tempdata(observables);

  double step = (var->upperlimit - var->lowerlimit) / var->numbins; 
  for (int i = 0; i < var->numbins; ++i) {
    var->value = var->lowerlimit + (i+0.5)*step;
    tempdata.addEvent(); 
  }
  setData(&tempdata);  
 
  thrust::counting_iterator<int> eventIndex(0); 
  thrust::constant_iterator<int> eventSize(observables.size()); 
  thrust::constant_iterator<fptype*> arrayAddress(cudaDataArray); 
  thrust::device_vector<fptype> results(var->numbins); 

  MetricTaker evalor(this, getMetricPointer("ptr_to_Eval")); 

#ifdef OMP_ON
  unsigned int thFirstEntry, thLastEntry;
  int tid, nthreads;

  tid = omp_get_thread_num();
  nthreads = omp_get_num_threads();

// use var->numbins or numEntries here?
  thFirstEntry = tid*(var->numbins)/nthreads;
  thLastEntry = (tid+1)*(var->numbins)/nthreads;

  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex+thFirstEntry, arrayAddress, eventSize)),
		    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + thLastEntry, arrayAddress, eventSize)),
		    results.begin()+thFirstEntry,
		    evalor); 
  #pragma omp barrier
#else
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
		    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
		    results.begin(),
		    evalor); 
#endif

  thrust::host_vector<fptype> h_results = results; 
  res.clear();
  res.resize(var->numbins);
  for (int i = 0; i < var->numbins; ++i) {
    res[i] = h_results[i] * host_normalisation[parameters];
  }
}

__host__ void ThrustPdfFunctor::evaluateAtPoints (std::vector<fptype>& points) const {
  /*
  std::set<Variable*> vars;
  getParameters(vars);
  unsigned int maxIndex = 0;
  for (std::set<Variable*>::iterator i = vars.begin(); i != vars.end(); ++i) {
    if ((*i)->getIndex() < maxIndex) continue;
    maxIndex = (*i)->getIndex();
  }
  std::vector<double> params;
  params.resize(maxIndex+1);
  for (std::set<Variable*>::iterator i = vars.begin(); i != vars.end(); ++i) {
    if (0 > (*i)->getIndex()) continue;
    params[(*i)->getIndex()] = (*i)->value;
  } 
  copyParams(params); 

  thrust::device_vector<fptype> d_vec = points; 
  normalise(); 
  hipMemcpyToSymbol(HIP_SYMBOL(normalisationFactors), host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 
  thrust::transform(d_vec.begin(), d_vec.end(), d_vec.begin(), *evalor);
  thrust::host_vector<fptype> h_vec = d_vec;
  for (unsigned int i = 0; i < points.size(); ++i) points[i] = h_vec[i]; 
  */
}

__host__ void ThrustPdfFunctor::scan (Variable* var, std::vector<fptype>& values) {
  fptype step = var->upperlimit;
  step -= var->lowerlimit;
  step /= var->numbins;
  values.clear(); 
  for (fptype v = var->lowerlimit + 0.5*step; v < var->upperlimit; v += step) {
    var->value = v;
    copyParams();
    fptype curr = calculateNLL(); 
    values.push_back(curr);
  }
}

__host__ void ThrustPdfFunctor::setParameterConstantness (bool constant) {
  std::set<Variable*> pars;
  getParameters(pars); 
  for (std::set<Variable*>::iterator p = pars.begin(); p != pars.end(); ++p) {
    (*p)->fixed = constant; 
  }
}

__host__ fptype ThrustPdfFunctor::getValue () {
  // Returns the value of the PDF at a single point. 
  // Execute redundantly in all threads for OpenMP multiGPU case
  copyParams(); 
  normalise(); 
  hipMemcpyToSymbol(HIP_SYMBOL(normalisationFactors), host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 

  UnbinnedDataSet point(observables); 
  point.addEvent(); 
  setData(&point); 

  thrust::counting_iterator<int> eventIndex(0); 
  thrust::constant_iterator<int> eventSize(observables.size()); 
  thrust::constant_iterator<fptype*> arrayAddress(cudaDataArray); 
  thrust::device_vector<fptype> results(1); 
  
  MetricTaker evalor(this, getMetricPointer("ptr_to_Eval"));
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
		    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + 1, arrayAddress, eventSize)),
		    results.begin(),
		    evalor); 
  return results[0];
}

__host__ fptype ThrustPdfFunctor::normalise () const {
  //if (cpuDebug & 1) std::cout << "Normalising " << getName() << " " << hasAnalyticIntegral() << " " << normRanges << std::endl;

  if (!fitControl->metricIsPdf()) {
    host_normalisation[parameters] = 1.0; 
    return 1.0;
  }

  fptype ret = 1;
  if (hasAnalyticIntegral()) {
    for (obsConstIter v = obsCBegin(); v != obsCEnd(); ++v) { // Loop goes only over observables of this PDF. 
      //std::cout << "Analytically integrating " << getName() << " over " << (*v)->name << std::endl; 
      ret *= integrate((*v)->lowerlimit, (*v)->upperlimit);
    }
    host_normalisation[parameters] = 1.0/ret;
    //if (cpuDebug & 1) std::cout << "Analytic integral of " << getName() << " is " << ret << std::endl; 
    return ret; 
  } 

  int totalBins = 1; 
  for (obsConstIter v = obsCBegin(); v != obsCEnd(); ++v) {
    ret *= ((*v)->upperlimit - (*v)->lowerlimit);
    totalBins *= (integrationBins > 0 ? integrationBins : (*v)->numbins); 
    //if (cpuDebug & 1) std::cout << "Total bins " << totalBins << " due to " << (*v)->name << " " << integrationBins << " " << (*v)->numbins << std::endl; 
  }
  ret /= totalBins; 

  fptype dummy = 0; 
  static thrust::plus<fptype> cudaPlus;
  thrust::constant_iterator<fptype*> arrayAddress(normRanges); 
  thrust::constant_iterator<int> eventSize(observables.size());
  thrust::counting_iterator<int> binIndex(0); 
#ifdef OMP_ON
  unsigned int thFirstBin, thLastBin;
  int tid, nthreads;
  int j;

  tid = omp_get_thread_num();
  nthreads = omp_get_num_threads();

  thFirstBin = tid*(totalBins)/nthreads;
  thLastBin = (tid+1)*(totalBins)/nthreads;
 
  //std::cout << "totalBins = " << totalBins << " thFirstBin = " << thFirstBin << " thLastBin = " << thLastBin << std::endl;

  sums[tid] = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(binIndex + thFirstBin, eventSize, arrayAddress)),
					thrust::make_zip_iterator(thrust::make_tuple(binIndex + thLastBin, eventSize, arrayAddress)),
					*logger, dummy, cudaPlus); 
  hipDeviceSynchronize(); // Ensure logger is done

  #pragma omp barrier
  if (tid == 0)
  {
    gSum = 0;
    for (j=0; j<nthreads; j++) gSum += sums[j];
  }
    
  //  std::cout << tid << ": sum = " << sum << " gSum = " << gSum << std::endl;

  #pragma omp barrier

  if (isnan(gSum)) {
    abortWithCudaPrintFlush(__FILE__, __LINE__, getName() + " NaN in normalisation", this); 
  }
  else if (0 == gSum) { 
    abortWithCudaPrintFlush(__FILE__, __LINE__, "Zero in normalisation", this); 
  }
 
  ret *= gSum;
#else
  fptype sum = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(binIndex, eventSize, arrayAddress)),
					thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, eventSize, arrayAddress)),
					*logger, dummy, cudaPlus); 
 
  if (isnan(sum)) {
    abortWithCudaPrintFlush(__FILE__, __LINE__, getName() + " NaN in normalisation", this); 
  }
  else if (0 == sum) { 
    abortWithCudaPrintFlush(__FILE__, __LINE__, "Zero in normalisation", this); 
  }

  //if (cpuDebug & 1) std::cout << getName() << " integral is " << ret << " " << sum << " " << (ret*sum) << " " << (1.0/(ret*sum)) << std::endl; 

  ret *= sum;
#endif

  if (0 == ret) abortWithCudaPrintFlush(__FILE__, __LINE__, "Zero integral"); 
  host_normalisation[parameters] = 1.0/ret;
  return (fptype) ret; 
}

// Notice that operators are distinguished by the order of the operands,
// and not otherwise! It's up to the user to make his tuples correctly. 

// Main operator: Calls the PDF to get a predicted value, then the metric 
// to get the goodness-of-prediction number which is returned to MINUIT. 
__device__ fptype MetricTaker::operator () (thrust::tuple<int, fptype*, int> t) const {
  int eventIndex = thrust::get<0>(t);
  int eventSize  = thrust::get<2>(t);
  fptype* eventAddress = thrust::get<1>(t) + (eventIndex * abs(eventSize)); 

  // Causes stack size to be statically undeterminable.
  fptype ret = (*(reinterpret_cast<device_function_ptr>(device_function_table[functionIdx])))(eventAddress, hipArray, paramIndices+parameters);

  // Notice assumption here! For unbinned fits the 'eventAddress' pointer won't be used
  // in the metric, so it doesn't matter what it is. For binned fits it is assumed that
  // the structure of the event is (obs1 obs2... binentry binvolume), so that the array
  // passed to the metric consists of (binentry binvolume); unless the data has 
  // user-provided errors, in which case binvolume is replaced by binError. 
  ret = (*(reinterpret_cast<device_metric_ptr>(device_function_table[metricIndex])))(ret, eventAddress + (abs(eventSize)-2), parameters);
  return ret; 
}
 
// Operator for binned evaluation, no metric. 
// Used in normalisation. 
#define MAX_NUM_OBSERVABLES 5
__device__ fptype MetricTaker::operator () (thrust::tuple<int, int, fptype*> t) const {
  // Bin index, event size, base address [lower, upper, numbins] 
 
  int evtSize = thrust::get<1>(t);
  assert(evtSize <= MAX_NUM_OBSERVABLES); 
  int binNumber = thrust::get<0>(t);
  
  // Do not understand why this cannot be declared __shared__. Dynamically allocating shared memory is apparently complicated. 
  //fptype* binCenters = (fptype*) malloc(evtSize * sizeof(fptype));
  __shared__ fptype binCenters[1024*MAX_NUM_OBSERVABLES];

  // To convert global bin number to (x,y,z...) coordinates: For each dimension, take the mod 
  // with the number of bins in that dimension. Then divide by the number of bins, in effect
  // collapsing so the grid has one fewer dimension. Rinse and repeat. 
  unsigned int* indices = paramIndices + parameters;
  for (int i = 0; i < evtSize; ++i) {
    fptype lowerBound = thrust::get<2>(t)[3*i+0];
    fptype upperBound = thrust::get<2>(t)[3*i+1];
    int numBins    = (int) FLOOR(thrust::get<2>(t)[3*i+2] + 0.5); 
    int localBin = binNumber % numBins;

    fptype x = upperBound - lowerBound; 
    x /= numBins;
    x *= (localBin + 0.5); 
    x += lowerBound;
    binCenters[indices[indices[0] + 2 + i]+threadIdx.x*MAX_NUM_OBSERVABLES] = x; 
    binNumber /= numBins;

    //if (gpuDebug & 1) 
    //if ((gpuDebug & 1) && (0 == threadIdx.x) && (0 == blockIdx.x)) 
      //printf("[%i, %i] Bins: %i %i %i %f %f %f %f %i\n", blockIdx.x, threadIdx.x, binNumber, numBins, localBin, x, lowerBound, upperBound, thrust::get<2>(t)[3*i+2], indices[indices[0] + 2 + i]); 
      //printf("Bins: %i %i %i %f %f\n", i, indices[indices[0] + 2 + i]+threadIdx.x*MAX_NUM_OBSERVABLES, indices[indices[0] + 2 + i], x, binCenters[threadIdx.x*MAX_NUM_OBSERVABLES]); 
  }

  // Causes stack size to be statically undeterminable.
  fptype ret = (*(reinterpret_cast<device_function_ptr>(device_function_table[functionIdx])))(binCenters+threadIdx.x*MAX_NUM_OBSERVABLES, hipArray, indices);
  //if (gpuDebug & 1) printf("[%i, %i] Binned eval: %f %f\n", blockIdx.x, threadIdx.x, binCenters[threadIdx.x*4], ret);
  return ret; 
}

__host__ void ThrustPdfFunctor::getCompProbsAtDataPoints (std::vector<std::vector<fptype> >& values) {
  //cpuDebug = 1; 
  copyParams(); 
  double overall = normalise();
  hipMemcpyToSymbol(HIP_SYMBOL(normalisationFactors), host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 
  //setDebugMask(1); 

  int numVars = observables.size(); 
  if (fitControl->binnedFit()) {
    numVars += 2;
    numVars *= -1; 
  }
  thrust::device_vector<fptype> results(numEntries); 
  thrust::constant_iterator<int> eventSize(numVars); 
  thrust::constant_iterator<fptype*> arrayAddress(cudaDataArray); 
  thrust::counting_iterator<int> eventIndex(0); 
  MetricTaker evalor(this, getMetricPointer("ptr_to_Prob")); 
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
		    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
		    results.begin(), 
		    evalor); 
  //setDebugMask(0); 
  values.clear(); 
  values.resize(components.size() + 1);
  thrust::host_vector<fptype> host_results = results;
  //std::cout << "Overall: " << overall << " " << host_normalisation[getParameterIndex()] << " " << host_results[0] << " " << numVars << " " << numEntries << " " << host_results.size() << std::endl; 
  for (unsigned int i = 0; i < host_results.size(); ++i) {
    values[0].push_back(host_results[i]);
  }
  
  for (unsigned int i = 0; i < components.size(); ++i) {
    MetricTaker compevalor(components[i], getMetricPointer("ptr_to_Prob")); 
    thrust::counting_iterator<int> ceventIndex(0); 
    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(ceventIndex, arrayAddress, eventSize)),
		      thrust::make_zip_iterator(thrust::make_tuple(ceventIndex + numEntries, arrayAddress, eventSize)),
		      results.begin(), 
		      compevalor); 
    host_results = results;
    //std::cout << "Normalisation " << components[i]->getName() << ": " << host_results[0] << ", " << host_normalisation[components[i]->getParameterIndex()] << std::endl; 
    for (unsigned int j = 0; j < host_results.size(); ++j) {
      values[1 + i].push_back(host_results[j]); 
    }
    
  }
}

// still need to add OpenMP/multi-GPU code here
__host__ void ThrustPdfFunctor::transformGrid (fptype* host_output) { 
  generateNormRange(); 
  //normalise(); 
  int totalBins = 1; 
  for (obsConstIter v = obsCBegin(); v != obsCEnd(); ++v) {
    totalBins *= (*v)->numbins; 
  }

  thrust::constant_iterator<fptype*> arrayAddress(normRanges); 
  thrust::constant_iterator<int> eventSize(observables.size());
  thrust::counting_iterator<int> binIndex(0); 
  thrust::device_vector<fptype> d_vec;
  d_vec.resize(totalBins); 

  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(binIndex, eventSize, arrayAddress)),
		    thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, eventSize, arrayAddress)),
		    d_vec.begin(), 
		    *logger); 

  thrust::host_vector<fptype> h_vec = d_vec;
  for (unsigned int i = 0; i < totalBins; ++i) host_output[i] = h_vec[i]; 
}

MetricTaker::MetricTaker (FunctorBase* dat, void* dev_functionPtr) 
  : metricIndex(0)
  , functionIdx(dat->getFunctionIndex())
  , parameters(dat->getParameterIndex())
{
  //std::cout << "MetricTaker constructor with " << functionIdx << std::endl; 

#ifdef OMP_ON
  int tid = omp_get_thread_num();
  std::map<void*, int>::iterator localPos = functionAddressToDeviceIndexMap[tid].find(dev_functionPtr); // Use find instead of [] to avoid returning 0 if the index doesn't exist.
  if (localPos != functionAddressToDeviceIndexMap[tid].end()) {
    metricIndex = (*localPos).second; 
  }
#else
  std::map<void*, int>::iterator localPos = functionAddressToDeviceIndexMap.find(dev_functionPtr); 
  if (localPos != functionAddressToDeviceIndexMap.end()) {
    metricIndex = (*localPos).second; 
  }
#endif
  else {
    metricIndex = num_device_functions; 
    host_function_table[num_device_functions] = dev_functionPtr;
#ifdef OMP_ON
    functionAddressToDeviceIndexMap[tid][dev_functionPtr] = num_device_functions; 
#else
    functionAddressToDeviceIndexMap[dev_functionPtr] = num_device_functions; 
#endif
    num_device_functions++; 
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(device_function_table), host_function_table, num_device_functions*sizeof(void*))); 
  }
}

MetricTaker::MetricTaker (int fIdx, int pIdx) 
  : metricIndex(0)
  , functionIdx(fIdx)
  , parameters(pIdx)
{
  // This constructor should only be used for binned evaluation, ie for integrals. 
}

__host__ void ThrustPdfFunctor::setFitControl (FitControl* const fc, bool takeOwnerShip) {
  for (unsigned int i = 0; i < components.size(); ++i) {
    components[i]->setFitControl(fc, false); 
  }

  if ((fitControl) && (fitControl->getOwner() == this)) {
    delete fitControl; 
  }
  fitControl = fc; 
  if (takeOwnerShip) {
    fitControl->setOwner(this); 
  }
  setMetrics();
}

#include "FunctorBase.cu" 
