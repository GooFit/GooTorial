#include "hip/hip_runtime.h"
#include "EventWeightedAddThrustFunctor.hh"

__device__ fptype device_EventWeightedAddPdfs (fptype* evt, fptype* p, unsigned int* indices) { 
  int numParameters = indices[0]; 
  fptype ret = 0;
  fptype totalWeight = 0; 

  for (int i = 0; i < numParameters/2 - 1; ++i) {
    fptype weight = evt[indices[2 + numParameters + i]];
    totalWeight += weight;
    fptype curr = (*(reinterpret_cast<device_function_ptr>(device_function_table[indices[2*i + 1]])))(evt, p, paramIndices + indices[2*(i+1)]);
    ret += weight * curr * normalisationFactors[indices[2*(i+1)]]; 
  }
  // numParameters does not count itself. So the array structure for two functions is
  // nP | F P | F P | nO | o1 
  // in which nP = 4. and nO = 1. Therefore the parameter index for the last function pointer is nP, and the function index is nP-1. 
  fptype last = (*(reinterpret_cast<device_function_ptr>(device_function_table[indices[numParameters-1]])))(evt, p, paramIndices + indices[numParameters]);
  ret += (1 - totalWeight) * last * normalisationFactors[indices[numParameters]]; 
  
  return ret; 
}

__device__ fptype device_EventWeightedAddPdfsExt (fptype* evt, fptype* p, unsigned int* indices) { 
  // numParameters does not count itself. So the array structure for two functions is
  // nP | F P | F P | nO | o1 o2
  // in which nP = 4, nO = 2. 

  int numParameters = indices[0]; 
  fptype ret = 0;
  fptype totalWeight = 0; 
  for (int i = 0; i < numParameters/2; ++i) {
    fptype curr = (*(reinterpret_cast<device_function_ptr>(device_function_table[indices[2*i + 1]])))(evt, p, paramIndices + indices[2*(i+1)]);
    fptype weight = evt[indices[2 + numParameters + i]];
    ret += weight * curr * normalisationFactors[indices[2*(i+1)]]; 
    totalWeight += weight; 

    //if ((gpuDebug & 1) && (0 == threadIdx.x))
    //if ((gpuDebug & 1) && (1 > evt[8]))
    //if ((gpuDebug & 1) && (0 == threadIdx.x) && (0 == blockIdx.x))
    //printf("EventWeightedExt: %i %f %f | %f %f %f %f %f %f %f\n", i, curr, weight, evt[0], evt[1], evt[2], evt[3], evt[4], evt[5], evt[6]);
    //printf("EventWeightedExt: %i %f %f | %f %f \n", i, curr, weight, normalisationFactors[indices[2*(i+1)]], curr * normalisationFactors[indices[2*(i+1)]]);
    //printf("EventWeightedExt: %i : %i %.10f %.10f %.10f %f %f %f\n", (int) floor(0.5 + evt[8]), i, curr, weight, ret, normalisationFactors[indices[2*(i+1)]], evt[6], evt[7]);
  }
  ret /= totalWeight; 
  
  return ret; 
}

__device__ device_function_ptr ptr_to_EventWeightedAddPdfs = device_EventWeightedAddPdfs; 
__device__ device_function_ptr ptr_to_EventWeightedAddPdfsExt = device_EventWeightedAddPdfsExt; 

EventWeightedAddThrustFunctor::EventWeightedAddThrustFunctor (std::string n, std::vector<Variable*> weights, std::vector<FunctorBase*> comps) 
  : ThrustPdfFunctor(0, n) 
{
  assert((weights.size() == comps.size()) || (weights.size() + 1 == comps.size())); 

  // Indices stores (function index)(function parameter index) doublet for each component. 
  // Last component has no weight index unless function is extended. Notice that in this case, unlike
  // AddThrustFunctor, weight indices are into the event, not the parameter vector, hence they
  // are not added to the pindices array at this stage, although 'initialise' will reserve space
  // for them. 
  for (std::vector<FunctorBase*>::iterator p = comps.begin(); p != comps.end(); ++p) {
    //std::cout << "EventWeighted component: " << (*p)->getName() << std::endl; 
    components.push_back(*p); 
    assert(components.back()); 
  }

  bool extended = true; 
  std::vector<unsigned int> pindices;
  for (unsigned int w = 0; w < weights.size(); ++w) {
    assert(components[w]);
    pindices.push_back(components[w]->getFunctionIndex());
    pindices.push_back(components[w]->getParameterIndex());
    registerObservable(weights[w]); 
  }
  assert(components.back()); 
  if (weights.size() < components.size()) {
    pindices.push_back(components.back()->getFunctionIndex());
    pindices.push_back(components.back()->getParameterIndex());
    extended = false; 
  }

  // This must occur after registering weights, or the indices will be off - the device functions assume that the weights are first. 
  getObservables(observables); 

  if (extended) hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_EventWeightedAddPdfsExt, sizeof(void*));
  else hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_EventWeightedAddPdfs, sizeof(void*));
  initialise(pindices);
} 

__host__ fptype EventWeightedAddThrustFunctor::normalise () const {
  //if (cpuDebug & 1) std::cout << "Normalising EventWeightedAddThrustFunctor " << getName() << " " << components.size() << std::endl;

  // Here the PDFs have per-event weights, so there is no per-PDF weight
  // to keep track of. All we can do is normalise the components. 
  for (unsigned int i = 0; i < components.size(); ++i) {
    components[i]->normalise(); 
  }
  host_normalisation[parameters] = 1.0; 

  return 1.0; 
}

