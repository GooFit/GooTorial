#include "hip/hip_runtime.h"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include "thrust/transform_reduce.h"
#include "rootstuff/TRandom.hh"
#include "rootstuff/TMinuit.hh" 
#include "cuPrintf.cu"

using namespace thrust; 

// Pointer to avoid crash on exit. 
device_vector<double>* dev_data;

// 512 functions should be enough for anyone.
__device__ void* dev_fcn_table[512];
__constant__ double dev_params[512]; 
__constant__ unsigned int dev_indices[512]; 
typedef double (*dev_fcn_ptr) (double, unsigned int); 


__device__ double dev_Gaussian (double xval, unsigned int pIdx) {
  double mean  = dev_params[dev_indices[pIdx + 1]]; // Not a typo
  double sigma = dev_params[dev_indices[pIdx + 2]]; 

  //printf("Gauss: %f %f %f %i %i\n", xval, mean, sigma, dev_indices[1], dev_indices[2]); 

  double ret = exp(-0.5*pow((xval - mean) / sigma, 2));
  ret /= sigma;
  ret /= sqrt(2*M_PI); 
  return ret;
}

__device__ double dev_BreitWigner (double xval, unsigned int pIdx) {
  double mean  = dev_params[dev_indices[pIdx + 1]];
  double width = dev_params[dev_indices[pIdx + 2]]; 

  double arg = xval - mean;
  double ret = width / (arg*arg + 0.25*width*width);
  ret /= (2*M_PI); // Normalise over -inf to inf 
  return ret; 
}

__device__ double dev_Polynomial (double xval, unsigned int pIdx) {
  // nP  c1  c2  c3  ...
  int numParams = dev_indices[pIdx]; 

  double ret = 0; 
  for (int i = 0; i < numParams; ++i) {
    double coef  = dev_params[dev_indices[pIdx + i + 1]];
    double power = pow(xval, i);
    ret         += coef*power; 
  }

  // Not good to normalise from -inf to inf.
  // Avoid the problem for now by hardcoding 
  // integration limits. Note use of numerical
  // integration to avoid places where actual
  // polynomial goes negative - PDF is always
  // positive! 
  double integral = 0; 
  for (double xint = -5.0; xint < 5.0; xint += 0.01) {
    double curr = 0; 
    for (int i = 0; i < numParams; ++i) {
      double coef  = dev_params[dev_indices[pIdx + i + 1]];
      double power = pow(xint, i);
      curr        += coef*power;
    }
    if (curr < 0) continue;
    integral += 0.01*curr; 
  }

  //printf("Poly: %f %f %f\n", xval, ret, integral);

  ret /= integral;
  return max(ret, 1e-6); 
}

__device__ double dev_SumOfFunctions (double xval, unsigned int pIdx) {
  double ret = 0; 
  
  unsigned int numPar = dev_indices[pIdx]; 
  unsigned int numFuncs = numPar / 3; // Actually number of functions minus one
  double totalWeight = 0; 
  for (unsigned int i = 0; i < numFuncs; ++i) {
    unsigned int currFuncStart = 1 + i*3; 
    double currWeight = dev_params[dev_indices[pIdx + currFuncStart]]; 
    totalWeight += currWeight; 

    unsigned int currFcnIdx = dev_indices[pIdx + currFuncStart + 1];
    unsigned int currParIdx = dev_indices[pIdx + currFuncStart + 2];
    dev_fcn_ptr theFunction;
    theFunction = reinterpret_cast<dev_fcn_ptr>(dev_fcn_table[currFcnIdx]); 
    double pdfVal = (*theFunction)(xval, currParIdx); 
    
    ret += pdfVal * currWeight;
  }

  // Last function has no separate weight. 
  unsigned int lastFcnIdx = dev_indices[pIdx + 1 + numFuncs * 3];
  unsigned int lastParIdx = dev_indices[pIdx + 2 + numFuncs * 3];
  dev_fcn_ptr theFunction;
  theFunction = reinterpret_cast<dev_fcn_ptr>(dev_fcn_table[lastFcnIdx]); 
  double pdfVal = (*theFunction)(xval, lastParIdx); 

  ret += pdfVal * (1 - totalWeight);
  return ret; 

}

__device__ dev_fcn_ptr ptr_to_Gaussian        = dev_Gaussian;
__device__ dev_fcn_ptr ptr_to_BreitWigner     = dev_BreitWigner;
__device__ dev_fcn_ptr ptr_to_Polynomial      = dev_Polynomial; 
__device__ dev_fcn_ptr ptr_to_SumOfFunctions  = dev_SumOfFunctions; 
int host_fcnIdx = 0; 

// Note that 'unary_function' is ambiguous with STL class of same name. 
struct GeneralFcn : public thrust::unary_function<double, double> {

  GeneralFcn (unsigned int idx, unsigned int pid) 
    : fcnIdx(idx)
    , parIdx(pid)
  {}

  __device__ double operator () (double xval) {
    dev_fcn_ptr theFunction;
    theFunction = reinterpret_cast<dev_fcn_ptr>(dev_fcn_table[fcnIdx]); 
    double pdfVal = (*theFunction)(xval, parIdx); 
    return -2*log(pdfVal);
  }
  
private:
  unsigned int fcnIdx;
  unsigned int parIdx; 
}; 

void fcn_glue (int& npar, double* deriv, double& fVal, double param[], int flag) {
  GeneralFcn functor(host_fcnIdx, 0); 
  double initVal = 0; 
  hipMemcpyToSymbol(HIP_SYMBOL(dev_params), param, npar*sizeof(double)); 
  fVal = transform_reduce(dev_data->begin(),
			  dev_data->end(), 
			  functor, 
			  initVal, 
			  thrust::plus<double>());
}

int main (int argc, char** argv) {
  // Generate random data
  TRandom donram(42); 
  host_vector<double> host_data;
  for (int i = 0; i < 100000; ++i) {
    double dieroll = donram.Uniform(); 
    if (dieroll < 0.2) host_data.push_back(donram.Gaus(0.0, 0.42));
    else host_data.push_back(donram.Gaus(0.0, 0.91));
  }
  // Move to device
  dev_data = new device_vector<double>(host_data); 
 
  // Initialise function table
  void* host_fcn_ptrs[512];
  hipMemcpyFromSymbol(host_fcn_ptrs+0, HIP_SYMBOL(ptr_to_Gaussian), sizeof(void*));
  hipMemcpyFromSymbol(host_fcn_ptrs+1, HIP_SYMBOL(ptr_to_BreitWigner), sizeof(void*));
  hipMemcpyFromSymbol(host_fcn_ptrs+2, HIP_SYMBOL(ptr_to_Polynomial), sizeof(void*));
  hipMemcpyFromSymbol(host_fcn_ptrs+3, HIP_SYMBOL(ptr_to_SumOfFunctions), sizeof(void*));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_fcn_table), host_fcn_ptrs, 4*sizeof(void*)); 

  host_fcnIdx = 3; 
  unsigned int npars = 4;
  TMinuit minuit(npars); 

  minuit.DefineParameter(0, "weight1",  0.30, 0.01, 0.0, 1.0); 
  minuit.DefineParameter(1, "mean",     0.30, 0.01, -10.0, 10.0); 
  minuit.DefineParameter(2, "sigma1",   0.20, 0.01, 0.01, 1.00); 
  minuit.DefineParameter(3, "sigma2",   0.70, 0.01, 0.20, 1.50); 

  unsigned int host_indices[12]; 
  host_indices[0] = 5; // Number of parameters for sum
  host_indices[1] = 0; // Index of weight parameter
  host_indices[2] = 0; // Index of Gaussian function
  host_indices[3] = 6; // Parameters for first Gaussian start at 6
  host_indices[4] = 0; // Gaussian function again
  host_indices[5] = 9; // Second parameter start

  // Conceptual break: Sum ends, first Gaussian starts

  host_indices[6] = 2; // Number of parameters for first Gaussian
  host_indices[7] = 1; // Index of mean parameter
  host_indices[8] = 2; // Index of first sigma

  // Another break: Second Gaussian starts

  host_indices[9] = 2; // Second Gaussian also has two parameters
  host_indices[10] = 1; // Mean is shared
  host_indices[11] = 3; // Second sigma

  hipMemcpyToSymbol(HIP_SYMBOL(dev_indices), host_indices, 12*sizeof(unsigned int)); 
  minuit.SetFCN(fcn_glue);
  minuit.Migrad(); 

  // Free the device memory. 
  delete dev_data; 
  return 0; 
}
