#include "hip/hip_runtime.h"
#include "Variable.hh" 
#include "GaussianThrustFunctor.hh" 
#include "ProdThrustFunctor.hh" 
#include "PdfBuilder.hh" 
#include "UnbinnedDataSet.hh" 

#include "TRandom.hh" 
#include "TH1F.h"
#include "TH2F.h"
#include "TStyle.h" 
#include "TCanvas.h" 

#include <sys/time.h>
#include <sys/times.h>
#include <iostream>

using namespace std; 

int main (int argc, char** argv) {
  gStyle->SetCanvasBorderMode(0);
  gStyle->SetCanvasColor(10);
  gStyle->SetFrameFillColor(10);
  gStyle->SetFrameBorderMode(0);
  gStyle->SetPadColor(0);
  gStyle->SetTitleColor(1);
  gStyle->SetStatColor(0);
  gStyle->SetFillColor(0);
  gStyle->SetFuncWidth(1);
  gStyle->SetLineWidth(1);
  gStyle->SetLineColor(1);
  gStyle->SetPalette(1, 0);

  vector<Variable*> vars; 
  Variable* xvar = new Variable("xvar", -5, 5); vars.push_back(xvar);
  Variable* yvar = new Variable("yvar", -5, 5); vars.push_back(yvar);
  UnbinnedDataSet data(vars);

  TH2F dataHist("dataHist", "", 
		xvar->numbins, xvar->lowerlimit, xvar->upperlimit,
		yvar->numbins, yvar->lowerlimit, yvar->upperlimit); 
  TH1F xvarHist("xvarHist", "", 
		xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  TH1F yvarHist("yvarHist", "", 
		yvar->numbins, yvar->lowerlimit, yvar->upperlimit);

  dataHist.SetStats(false); 
  xvarHist.SetStats(false); 
  yvarHist.SetStats(false); 

  TRandom donram(42); 
  double totalData = 0; 
  for (int i = 0; i < 100000; ++i) {
    xvar->value = donram.Gaus(0.2, 1.1);
    yvar->value = donram.Gaus(0.5, 0.3);
    if (fabs(xvar->value) > 5) {--i; continue;} 
    if (fabs(yvar->value) > 5) {--i; continue;} 
    data.addEvent(); 
    dataHist.Fill(xvar->value, yvar->value); 
    xvarHist.Fill(xvar->value);
    yvarHist.Fill(yvar->value);
    totalData++; 
  }

  Variable* xmean = new Variable("xmean", 0, 1, -10, 10);
  Variable* xsigm = new Variable("xsigm", 1, 0.5, 1.5); 
  GaussianThrustFunctor xgauss("xgauss", xvar, xmean, xsigm); 

  Variable* ymean = new Variable("ymean", 0, 1, -10, 10);
  Variable* ysigm = new Variable("ysigm", 0.4, 0.1, 0.6); 
  GaussianThrustFunctor ygauss("ygauss", yvar, ymean, ysigm); 

  vector<FunctorBase*> comps;
  comps.push_back(&xgauss);
  comps.push_back(&ygauss);

  ProdThrustFunctor total("total", comps);
  total.setData(&data);
  PdfFunctor fitter(&total);
  fitter.fit(); 
  fitter.getMinuitValues(); 
 
  TH2F pdfHist("pdfHist", "", 
	       xvar->numbins, xvar->lowerlimit, xvar->upperlimit,
	       yvar->numbins, yvar->lowerlimit, yvar->upperlimit); 
  TH1F xpdfHist("xpdfHist", "", 
		xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  TH1F ypdfHist("ypdfHist", "", 
		yvar->numbins, yvar->lowerlimit, yvar->upperlimit);

  pdfHist.SetStats(false);
  xpdfHist.SetStats(false);
  ypdfHist.SetStats(false);

  UnbinnedDataSet grid(vars);
  for (int i = 0; i < xvar->numbins; ++i) {
    double step = (xvar->upperlimit - xvar->lowerlimit)/xvar->numbins;
    xvar->value = xvar->lowerlimit + (i + 0.5) * step;
    for (int j = 0; j < yvar->numbins; ++j) {
      step = (yvar->upperlimit - yvar->lowerlimit)/yvar->numbins;
      yvar->value = yvar->lowerlimit + (j + 0.5) * step;
      grid.addEvent(); 
    }
  }

  total.setData(&grid);
  vector<vector<double> > pdfVals;
  total.getCompProbsAtDataPoints(pdfVals); 

  TCanvas foo;
  dataHist.Draw("colz");
  foo.SaveAs("data.png"); 

  double totalPdf = 0; 
  for (int i = 0; i < grid.getNumEvents(); ++i) {
    grid.loadEvent(i); 
    pdfHist.Fill(xvar->value, yvar->value, pdfVals[0][i]);
    xpdfHist.Fill(xvar->value, pdfVals[0][i]);
    ypdfHist.Fill(yvar->value, pdfVals[0][i]);
    totalPdf += pdfVals[0][i]; 
  }

  for (int i = 0; i < xvar->numbins; ++i) {
    double val = xpdfHist.GetBinContent(i+1); 
    val /= totalPdf; 
    val *= totalData;
    xpdfHist.SetBinContent(i+1, val); 
  }
  for (int i = 0; i < yvar->numbins; ++i) {
    double val = ypdfHist.GetBinContent(i+1); 
    val /= totalPdf; 
    val *= totalData;
    ypdfHist.SetBinContent(i+1, val); 
    for (int j = 0; j < xvar->numbins; ++j) {
      val = pdfHist.GetBinContent(j+1, i+1); 
      val /= totalPdf; 
      val *= totalData;
      pdfHist.SetBinContent(j+1, i+1, val); 
    }
  }

  pdfHist.Draw("colz"); 
  foo.SaveAs("pdf.png"); 

  for (int i = 0; i < yvar->numbins; ++i) {
    for (int j = 0; j < xvar->numbins; ++j) {
      double pval = pdfHist.GetBinContent(j+1, i+1); 
      double dval = dataHist.GetBinContent(j+1, i+1); 
      pval -= dval;
      pval /= std::max(1.0, sqrt(dval));
      pdfHist.SetBinContent(j+1, i+1, pval); 
    }
  }

  pdfHist.GetZaxis()->SetRangeUser(-5, 5);
  pdfHist.Draw("colz"); 
  foo.SaveAs("pull.png"); 

  xvarHist.SetMarkerStyle(8);
  xvarHist.SetMarkerSize(0.5);
  xvarHist.Draw("p"); 
  xpdfHist.SetLineColor(kBlue);
  xpdfHist.SetLineWidth(3); 
  xpdfHist.Draw("lsame"); 
  foo.SaveAs("xhist.png"); 

  yvarHist.SetMarkerStyle(8);
  yvarHist.SetMarkerSize(0.5);
  yvarHist.Draw("p"); 
  ypdfHist.SetLineColor(kBlue);
  ypdfHist.SetLineWidth(3); 
  ypdfHist.Draw("lsame"); 
  foo.SaveAs("yhist.png"); 

  
  return 0;
}
